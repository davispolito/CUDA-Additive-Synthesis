#include "hip/hip_runtime.h"
#include "kernel.h"
#include <math.h>
#include <hip/device_functions.h>
#include <cmath>
#define M_PI 3.1415926535897931
#define THREADS_PER_SAMPLE 8
#define SAMPLES_PER_THREAD 10
#define ATOMIC_SYNTH 0
float* dev_frequencies, *dev_buffer;
int numSamples, numSinusoids;

void Additive::initSynth(int numSinusoid, int numSample, float* host_frequencies) {
	numSamples = numSample;
	numSinusoids = numSinusoid;
	hipMalloc((void**)&dev_frequencies, numSinusoids * sizeof(float));
	hipMalloc((void**)&dev_buffer, numSamples * sizeof(float));
	hipMemcpy(dev_frequencies, host_frequencies, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}

void Additive::endSynth() {
	hipFree(dev_buffer);
	hipFree(dev_frequencies);
}
__global__ void sin_kernel_simple(float * buffer, float* frequencies, float angle, int numSamples, int numSinusoids) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		angle = angle + 2.0f * M_PI * idx / 44100;
		float val = 0.0f;
		for (int i = 0; i < numSinusoids; i++){
			val += 0.01 * __sinf((angle * frequencies[i]));
			}
		buffer[idx] = val;

	}
}

void Additive::compute_sinusoid_gpu_simple(float* buffer, int angle) {
	int threadsPerBlock = 256; 
	int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;

	sin_kernel_simple << <blocksPerGrid, threadsPerBlock >> > (dev_buffer, dev_frequencies, angle, numSamples, numSinusoids);
	
	hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}

